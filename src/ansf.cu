#include "hip/hip_runtime.h"
#include<sys/mman.h>
#include<assert.h>

#include<iostream>
#include<string.h>
#include "../common.h"
#include "types.h"

//#include<unordered_map>
//#include<hip/hip_runtime.h>

// --- Add Constant Memory for Line Range ---
__constant__ int d_targetStartLine = -1; // Default: -1 means measure all lines
__constant__ int d_targetEndLine = -1;

__device__ int CTALB = 0; 		//the lower bound of CTA id you want to profile
__device__ int CTAUB = 99999;		//the upper bound of CTA id you want to profile 
__device__ int CONSTANCE = 128;
__device__ int aliveCTA = 0;

//__device__ std::unordered_map< std::string, long> blockmap;
//__device__ std::vector<int> testt;  		//this DOESN'T work //"dynamic initialization doesn't work for __device__

__device__ bool VERBOSE=false;
__device__ bool CALLPATHVERBOSE=false;

extern "C"
{	//so that no mangling for function names
	__device__ void takeString(void* , int);
	__device__ void RetKernel(void*);
	__device__ void passBasicBlock(int, int, int, int, void*);
	__device__ void print5(void*, int, int, int, int, void*);
	__device__ void print4(void*);
	__device__ void callFunc(void* , void* , int , int, void*);
	__device__ int getContextID(void*);
	__device__ void* InitKernel(void*);
	__device__ void print1(int);
}

__device__ unsigned long long ccnntt = 1;		//the very first element is reserved for metadata
__device__ unsigned long long bbccnntt = 1;		//the very first element is reserved for metadata

__device__ int* buffer_oN_DeViCe;		//should be multiples of 6
//__device__ int* globalCallStack;
//__device__ CallSite_t* globalCallStack;
//__device__ int* stackHeight;
 

__device__ char funcDic[UNIQUE_FUNC_DEVICE][FUNC_NAME_LEN];       //maintains 100 unique functions and 31 chars for each
__device__ int dicHeight = 0;				// size of funcDic[][]

__device__ CallSite_t contextDic[TOTAL_NUMBER_CONTEXT][CALL_PATH_LEN_DEVICE]; //maintains 100 unique contexts, each has up to 10 function
__device__ int cHeight = 0;



__device__ void mystrcpy(char* dst, char* src)
{
	int cnt = 0;
	while ( src[cnt] != '\0' && cnt < FUNC_NAME_LEN-1) //never exceeds this 30 limit
	{
		dst[cnt] = src[cnt];
		cnt++;
	}
	dst[cnt] = '\0';
	return;
}

__device__ bool mystrcmp(char* dst, char* src)
{
        int cnt = 0;
        while ( cnt < FUNC_NAME_LEN-1 ) //never exceeds this 30 limit
        {
		if ( dst[cnt] == '\0' && src[cnt] == '\0')
			return true;

                if (dst[cnt] != src[cnt])
			return false;
                cnt++;
        }
        return true;
}

__device__ int getFuncID(char* func)
{
	if (dicHeight == 0 ) //the very first function
	{
		mystrcpy(funcDic[0], func);
//		printf("src: %s\n", func);
//		printf("dst: %s\n", funcDic[0]);
		dicHeight ++;
		return 0;
	}

//	printf("d: height = %d\n", dicHeight);
	for(int i=0; i < dicHeight; i++)
	{
		bool found = mystrcmp( funcDic[i],  func );
//		printf("d:: compare this pair: %s: \t%s \tVS\t %s\n", found?"yes":"no", funcDic[i], func);
		if(found)
			return i;
	}

	//return -1;//DEBUG	
	//if you are here, means we have a new func
	mystrcpy(funcDic[dicHeight], func); 
	dicHeight ++;
	return dicHeight-1;
}

__device__ void updateCallStack(int caller, int callee, short sline, short scolm, int bid, int tid, void* p_stackzone)
{
	int offset = bid*blockDim.x*blockDim.y+tid;
//	CallSite_t* callStack = (CallSite_t*) (&(globalCallStack[offset*CALL_PATH_LEN_DEVICE]));
//	int &height = stackHeight[offset];

	CallSite_t* callStack = (CallSite_t*) p_stackzone;
	int bytesPerThread = (CALL_PATH_LEN_DEVICE*sizeof(CallSite_t));
	int* temp = (int*)( (char*)p_stackzone + bytesPerThread+16);	//offset by 16 to be safe, need to be consistent
	int &height = *temp;		

//	int &h11 = * (int*)( (char*)p_stackzone + bytesPerThread);		
	
//	if(CALLPATHVERBOSE)
//		printf( ":::::::: height = %d :::::::::\n", height);

//	assert(height != 1 && "stack height != 1") ;

	//return;//DUBUG
        if (height==0)
        {
//		if (CALLPATHVERBOSE)
//			printf("first ever. tid=%d\n", tid);
		callStack[0].id = caller;
		callStack[0].sline = sline;
		callStack[0].scolm = scolm;
		
		callStack[1].id = callee;
                callStack[1].sline = -1;
                callStack[1].scolm = -1;
		height=2;
                return;
        }

        int p_caller = callStack[height-2].id;
        int p_callee = callStack[height-1].id;

        if ( p_caller == caller && p_callee == callee)
        {       //repeated call
//		if (CALLPATHVERBOSE)
//			printf("repeated call\n");
                callStack[height-2].sline = sline;
                callStack[height-2].scolm = scolm;
                return;
        }
        else if ( p_caller == caller && p_callee != callee)
        {       //the same parent called a different function, simply update the callee
//		if (CALLPATHVERBOSE)
//      		printf("same caller different callee\n");
                callStack[height-1].id = callee;
                callStack[height-2].sline = sline;
                callStack[height-2].scolm = scolm;
                return;
        }
        else if ( p_callee == caller)
        {       // a typical call path
//		if (CALLPATHVERBOSE)
//			printf("call sequence\n");
                callStack[height-1].sline = sline;
                callStack[height-1].scolm = scolm;

                callStack[height].id = callee;
                callStack[height].sline = -1;
                callStack[height].scolm = -1;
		height++;
                return;
        }
//	return;//DUBUG

//	if (CALLPATHVERBOSE)
//		printf("the caller exists deeply in the stack\n");
	// the caller exists deeply in the stack
       	for (int i=height-1; i>=0; i--)
        {
                if ( callStack[i].id == caller)
                {
			height = i+1;
                        callStack[i].id = callee;
                        callStack[i].sline = -1;
                        callStack[i].scolm = -1;

                        callStack[i].sline = sline;
                        callStack[i].scolm = scolm;
                        return;
                }
        }

        // the caller exists deeply in the stack
//	assert( (0==-1) && "!! undefined things happeened here\n");
}



__device__ void* InitKernel(void* ptrhead)
{
	//TODO:
	if ( (blockIdx.x + blockIdx.y*gridDim.x) < CTALB || (blockIdx.x + blockIdx.y*gridDim.x) > CTAUB) // you only need a few CTAs
		return NULL; 

        int tid = threadIdx.x + threadIdx.y *blockDim.x;
        int bid = blockIdx.x + blockIdx.y * gridDim.x;
        int global_tid = tid + bid*blockDim.x*blockDim.y;
	int num_cta = gridDim.x*gridDim.y;
	int num_thread = blockDim.x*blockDim.y;

	__shared__ char* handler;	//this pointer is for maintaing stack/callpath
	__syncthreads();
	int bytesPerThread = sizeof(CallSite_t)*CALL_PATH_LEN_DEVICE + 32;// I put 32 just to be safe
	if ( tid ==0 )
	{
		handler = (char*) malloc( blockDim.x*blockDim.y*bytesPerThread); 
		assert( handler!=NULL);
//		printf(" CTA \t%d\tgrabs memroy\t%p\n", bid, handler);
		int rank = atomicAdd( &aliveCTA, 1);
		printf(" CTA\t%d\tonline, total alive\t%d\n", bid, rank);
		if (rank==0)
		{
		//	if (tid%32==0)
			{
		//		buffer_oN_DeViCe = (int*)ptrhead;
				printf("\nd: InitKernel...\n");
				printf("d: buffer pointer: %p\n", buffer_oN_DeViCe);
				printf("d: size of kernel grid: %d, %d\t%d, %d\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);
			}
		}
		if (rank == 1)
			buffer_oN_DeViCe = (int*)ptrhead;
	}	
	__syncthreads();

	void* stackzone = (void*)( handler + bytesPerThread*tid );

	return stackzone;


} 

__device__ void callFunc(void* er, void* ee, int sline, int scolm, void* p_stackzone)
{
	if (p_stackzone == NULL)
		return;
//	if (threadIdx.x != 0 || blockIdx.x != 0 || threadIdx.y != 0 || blockIdx.y != 0) 	return;	//DEBUG

//	printf("d::%d\n", sline );
//	printf("d::%s\n", (char*)er );

//	if (CALLPATHVERBOSE)
//		printf("d:::: >>>>\n");
	int id1 = getFuncID( (char*)er );
	int id2 = getFuncID( (char*)ee );
//	if (CALLPATHVERBOSE)
//	{
//		printf("d:::: ID: %d :%s\n", id1, (char*)er );
//		printf("d:::: ID: %d :%s\n", id2, (char*)ee );
//	}

	int tid = threadIdx.y * blockDim.x + threadIdx.x;
        int bid = blockIdx.x + blockIdx.y * gridDim.x;
        int global_tid = bid * (blockDim.x * blockDim.y) + tid;

	updateCallStack(id1, id2, (short) sline, (short) scolm, bid, tid, p_stackzone);
//	printCallStack(global_tid);

//	if (CALLPATHVERBOSE)
//		printf("d:::: <<<<\n");
}



__device__ void cxtprint(int id)
{
	if (id<0)
		return;

	printf("d::: requested context id: %d out of %d\n", id, cHeight);

	for (int i = 0; i< CALL_PATH_LEN_DEVICE  && contextDic[id][i].id != -1  ; i++)
	{
		printf("d::::::: current context [%d][%d]: %d, %d, %d\n", id, i, contextDic[id][i].id, contextDic[id][i].sline, contextDic[id][i].scolm) ;
	}

	return;
}

__device__ void cxtcpy( CallSite_t* dst, CallSite_t* src , int height) //context copy 
{

	int i;
	for( i=0; i< height; i++)
		dst[i] = src[i];

//	assert(i<CALL_PATH_LEN_DEVICE && "code: e56: call stack too deep");

	dst[i].id = -1; //to mark the ending of one context

	return;
}


__device__ bool cxtcmp( CallSite_t* dst, CallSite_t* src, int height)
{
	for( int i=0; i< height; i++)
		if ( dst[i].id == src[i].id ) // && dst[i].id == src[i].id && 	
			continue;
		else
			return false;

	return true;
}


__device__ int getContextID(void* p_stackzone)
{	//shared by all treahds, there are races
	//you can manually to take care of serialization?

//	if (threadIdx.x + threadIdx.y + blockIdx.x + blockIdx.y != 0 ) 		return -2; //DEBUG

	int bid = blockIdx.x + blockIdx.y * gridDim.x; 
	int tid = threadIdx.y * blockDim.x + threadIdx.x;

//	int offset = bid*blockDim.x*blockDim.y+tid;
//	CallSite_t* callStack = (CallSite_t*) (&(globalCallStack[offset*CALL_PATH_LEN_DEVICE]));
//      int &height = stackHeight[offset];

        CallSite_t* callStack = (CallSite_t*) p_stackzone;
	int bytesPerThread = (CALL_PATH_LEN_DEVICE*sizeof(CallSite_t));
	int* temp = (int*)( (char*)p_stackzone + bytesPerThread+16);	//offset by 8 to be safe, need to be consistent
	int &height = *temp;

	if ( height ==0) //it is possible that call stack is still empty
		return -1;

	if (cHeight==0)// the first ever context in the dic
	{
//		if (CALLPATHVERBOSE)
//			printf("d::: the very first context in dic, depth=%d\n", height);
		cxtcpy(contextDic[0], callStack, height );
		cHeight=1;
		return 0;
	}

	// something already exists
//	if (CALLPATHVERBOSE)
//	{
//		printf("d::: going to match existing items in context dic\n");
//		printf("d::: number of existing contexts: %d\n", cHeight);
//	}
	int i;
	for (i = 0; i<cHeight; i++)
	{
		if ( cxtcmp(  contextDic[i], callStack, height ) ) //yes, found
		{
//			if (CALLPATHVERBOSE)
//				printf("d::: matched, returning %d,  depth=%d\n",i, height);
			return i; 
		}
	}

//	if (CALLPATHVERBOSE)
//		printf("d::: not found, value of i: %d\n", i);

//	assert (i< TOTAL_NUMBER_CONTEXT && "code:e34: Not enough space for Context Dic, index i");
//	printCallStack();

	cxtcpy(contextDic[i], callStack, height );
	cHeight = i+1;
//	assert (cHeight < TOTAL_NUMBER_CONTEXT && "code:e41: Not enough space for Context Dic, cHeight");
	
//	if (CALLPATHVERBOSE)
//		printf("d::: inserted new one: id = %d, depth=%d\n", i, height);
	return i;	
}


__device__ void passBasicBlock(int tmp /*pointer to block name*/, int action, int sline, int scolm, void* p_stackzone)
{
        if ( (blockIdx.x + blockIdx.y*gridDim.x) < CTALB || (blockIdx.x + blockIdx.y*gridDim.x) > CTAUB) // you only need a few CTAs
                return;

	int map = __ballot_sync(0xFFFFFFFF, 1);
	int numActive = __popc(map);

	if ( buffer_oN_DeViCe == NULL)
		return;


	if (numActive==32)
	{
		//then choose one thread to write numbers
		int tid = threadIdx.x + threadIdx.y *blockDim.x;
		if (tid%32==0)
		{	
			//do the writing
			//	printf("I will write for my warp tid=(%d, %d)\n", threadIdx.x, threadIdx.y);
			int bid = atomicAdd(&bbccnntt, 1);
			unsigned long long key=0;

			BBlog_t* bblog = (BBlog_t*) buffer_oN_DeViCe;
			bblog[bid].key = key;

			bblog[bid].tidx = (short)threadIdx.x;
			bblog[bid].tidy = (short)threadIdx.y;
			bblog[bid].bidx = (short)blockIdx.x;
			bblog[bid].bidy = (short)blockIdx.y;
			bblog[bid].sline = sline;
			bblog[bid].scolm = scolm;

			bblog[bid].cid = getContextID(p_stackzone);
		}

	} 
	else
	{
		//every thread needs to write
		//printf("I will write for my self tid=(%d, %d)\n", threadIdx.x, threadIdx.y);
                        int bid = atomicAdd(&bbccnntt, 1);
                        unsigned long long key=0;

                        BBlog_t* bblog = (BBlog_t*) buffer_oN_DeViCe;
			bblog[bid].key = key;

                        bblog[bid].tidx = (short)threadIdx.x;
                        bblog[bid].tidy = (short)threadIdx.y;
                        bblog[bid].bidx = (short)blockIdx.x;
                        bblog[bid].bidy = (short)blockIdx.y;
                        bblog[bid].sline = sline;
                        bblog[bid].scolm = scolm;

                        bblog[bid].cid = getContextID(p_stackzone);
	}

        return;
}



__device__ void storeLines(void* p, short size/*bytes*/, short line, short colmn, short op /*load or store*/, void* p_stackzone)
{
        if ( (blockIdx.x + blockIdx.y*gridDim.x) < CTALB || (blockIdx.x + blockIdx.y*gridDim.x) > CTAUB) // you only need a few CTAs
                return;

        int map = __ballot_sync(0xFFFFFFFF, 1);
        int numActive = __popc(map);

	if ( ccnntt >  (int)(((long)BUFFERSIZE)/24) - 128*100)
		return; //DEBUG
	
	assert ( (ccnntt < BUFFERSIZE/24 - 128) && "code: e31: too many entries to the buffer"); //DO NOT COMMENT OUT
	
	int bid = atomicAdd(&ccnntt, 1);

	//d_trace[bid].bidx = blockIdx.x;
	//d_trace[bid].tidx = threadIdx.x;
	//d_trace[bid].ea = p;
	//d_trace[bid].bytes = size;
	//printf(" d : bid = %d from (%d,%d) (%d,%d) \n", bid, blockIdx.x, threadIdx.x, blockIdx.y, threadIdx.y);

	if (buffer_oN_DeViCe==NULL)
		return;

	if( true)
	{
		int tid = threadIdx.x + threadIdx.y *blockDim.x;
		if ( tid%32==0 || true)
		{

			short* buffer_oN_DeViCe_short = (short*) buffer_oN_DeViCe;
			long* buffer_oN_DeViCe_long = (long*) buffer_oN_DeViCe;

			buffer_oN_DeViCe_short[bid*12+0] = (short)blockIdx.x;
			buffer_oN_DeViCe_short[bid*12+1] = (short)blockIdx.y;
			buffer_oN_DeViCe_short[bid*12+2] = (short)threadIdx.x;
			buffer_oN_DeViCe_short[bid*12+3] = (short)threadIdx.y;
			buffer_oN_DeViCe_long[bid*3+1] = (long)p;
			buffer_oN_DeViCe_short[bid*12+8] = size;
			buffer_oN_DeViCe_short[bid*12+9] = line;
			buffer_oN_DeViCe_short[bid*12+10] = colmn;
			buffer_oN_DeViCe_short[bid*12+11] = op;
			getContextID(p_stackzone);
		}	
	}
}



__device__ void print1(int a)
{
	if (threadIdx.x + threadIdx.y + blockIdx.x + blockIdx.y == 0)
		printf("d: print1: %d\n", a);

	return;
	if (threadIdx.x + threadIdx.y + blockIdx.x + blockIdx.y == 0 && VERBOSE)
	{	
		if (a==1)
			printf("d: load by CTA (%d,%d)\n", blockIdx.x, blockIdx.y);
		else if (a==2)
			printf("d: store by CTA (%d,%d)\n", blockIdx.x, blockIdx.y);
		else
			printf("d: !!! undefined !!! \n" );
	}	
}

/*
__device__ void print2()
{
	if (threadIdx.x + threadIdx.y + blockIdx.x + blockIdx.y == 0 && VERBOSE)
        	printf("d: store by CTA (%d,%d)\n", blockIdx.x, blockIdx.y);
}
*/

__device__ void print3(int line, int col)
{
	return;
	if (threadIdx.x + threadIdx.y + blockIdx.x + blockIdx.y == 0 && VERBOSE)
        	printf("d: source line: %d\t column: %d by CTA (%d,%d)\n", line, col, blockIdx.x, blockIdx.y);
}

__device__ void print4(void* p)
{
        //if (threadIdx.x + threadIdx.y + blockIdx.x + blockIdx.y == 0 && VERBOSE)

	printf("d: print4: %p\n", p);

}

__device__ void print5(void* p, int bits, int sline, int scolm, int op, void* p_stackzone)
{
//	if ( (blockIdx.x  + blockIdx.y* gridDim.x) * (blockDim.x * blockDim.y) >= 32*128) // no more than 128 warps
//		return;
//	printf("d: ea: %p by (%d,%d) (%d,%d), CTA id = %d\n",p, blockIdx.x, threadIdx.x, blockIdx.y, threadIdx.y , (blockIdx.x  + blockIdx.y* gridDim.x));

	if ( (blockIdx.x + blockIdx.y*gridDim.x) < CTALB || (blockIdx.x + blockIdx.y*gridDim.x) > CTAUB) // you only need a few CTAs 
		return;

			bool isInRange = (d_targetStartLine == -1 || d_targetEndLine == -1 || (sline >= d_targetStartLine && sline <= d_targetEndLine));
			if (isInRange)
			{return;}
        storeLines(p, (short)(bits/8), (short)sline, (short) scolm, (short)op, p_stackzone);
//	printf("d: ea: %p by (%d,%d) (%d,%d), CTA id = %d\n",p, blockIdx.x, threadIdx.x, blockIdx.y, threadIdx.y , (blockIdx.x  + blockIdx.y* gridDim.x));
//	printf("d: ea: %p by (%d,%d) (%d,%d)\n",p, blockIdx.x, threadIdx.x, blockIdx.y, threadIdx.y );

}

////
__device__ void RetKernel(void* p_stackzone)
{
	if ( (blockIdx.x + blockIdx.y*gridDim.x) < CTALB || (blockIdx.x + blockIdx.y*gridDim.x) > CTAUB) // you only need a few CTAs 
		return;

        int bid = blockIdx.x + blockIdx.y * gridDim.x;
        int tid = threadIdx.x + threadIdx.y *blockDim.x;


	__syncthreads();	//IMPORTANT to sync here
	int rank = -1;
	if ( tid == 0)
	{
	//	printf(" CTA\t%d\treleases:\t%p\n", bid, stackzone);
	//	atomicAdd( &alive, -1);
		if (p_stackzone!=NULL)
		{
			free(p_stackzone);
			rank = atomicAdd( &aliveCTA, -1);
			printf("CTA\t%d\texits, total remains\t%d\n", bid, rank);
		}
		else
			printf("d:: p_stack is hacked!!\n");
	}
	__syncthreads();

	if (threadIdx.x + threadIdx.y == 0 && rank ==1 )
	{
		printf("d: in RetKernel...\n");

//                for (int kk=0; kk< cHeight; kk++)
//                        cxtprint( kk );

		if (true)
		{	//memory
			short* buffer_oN_DeViCe_short = (short*) buffer_oN_DeViCe;
			buffer_oN_DeViCe_short[0+0] = blockDim.x; // Be consistent with print.cpp, dumpTrace()
			buffer_oN_DeViCe_short[0+1] = blockDim.y;
			buffer_oN_DeViCe_short[0+2] = gridDim.x;
			buffer_oN_DeViCe_short[0+3] = gridDim.y;
			printf("d: Kernel Returns: collected [ %llu ] memory entries. \n" , ccnntt);
			printf("d: Kernel Returns: collected [ %llu ] memory entries. \n" , bbccnntt);

			long* buffer_oN_DeViCe_long = (long*) buffer_oN_DeViCe;
			buffer_oN_DeViCe_long[0+1] = ccnntt;
		}
		else
		{	//branch
			BBlog_t* bbbuffer_oN_DeViCe_short = (BBlog_t*) buffer_oN_DeViCe;
			bbbuffer_oN_DeViCe_short[0].bidx = blockDim.x; // Be consistent with print.cpp, dumpTrace()
			bbbuffer_oN_DeViCe_short[0].bidy = blockDim.y;
			bbbuffer_oN_DeViCe_short[0].tidx = gridDim.x;
			bbbuffer_oN_DeViCe_short[0].tidy = gridDim.y;
			bbbuffer_oN_DeViCe_short[0].key = bbccnntt;
			bbbuffer_oN_DeViCe_short[0].sline = 0;
			bbbuffer_oN_DeViCe_short[0].scolm = 0;
			printf("d: Kernel Returns: collected [ %llu ] BB logs. \n" , bbccnntt);
			printf("d: Kernel Returns: collected [ %llu ] BB logs. \n" , ccnntt);
		}

		unsigned long offset1 = ((UNIQUE_FUNC_DEVICE* FUNC_NAME_LEN*sizeof(char))/1024+1)*1024;
		unsigned long offset2 = ((TOTAL_NUMBER_CONTEXT * CALL_PATH_LEN_DEVICE* sizeof(CallSite_t))/1024+1)*1024 + offset1;

		printf("size of function dic: %d %d %lu -> %lu , rounded to %lu\n", UNIQUE_FUNC_DEVICE, FUNC_NAME_LEN, sizeof(char), UNIQUE_FUNC_DEVICE*FUNC_NAME_LEN*sizeof(char), offset1 );

                printf("size of context dic: %d %d %lu -> %lu , rounded to %lu\n", TOTAL_NUMBER_CONTEXT, CALL_PATH_LEN_DEVICE, sizeof(CallSite_t), TOTAL_NUMBER_CONTEXT* CALL_PATH_LEN_DEVICE* sizeof(CallSite_t) , offset2);


		//function dic is the last, 
		//context dic is second to last
		void* ptr;
		ptr = (void*)( buffer_oN_DeViCe + (BUFFERSIZE - offset1)/sizeof(int)) ; //operate on a int*, not a void*

		memcpy( ptr, funcDic, UNIQUE_FUNC_DEVICE *FUNC_NAME_LEN*sizeof(char) );

		ptr = (void*)(buffer_oN_DeViCe + (BUFFERSIZE - offset2)/sizeof(int)) ; //operate on a int*, not a void*
		memcpy( ptr, contextDic, TOTAL_NUMBER_CONTEXT * CALL_PATH_LEN_DEVICE*sizeof(CallSite_t) );
		ccnntt = 1; //reset, prepares for next kernel call
		bbccnntt = 1; //reset, prepares for next kernel call

	}//end of if

        

}

